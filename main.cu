/*
 *  Parallel Implementaion of Hough Transform for Circle Detection in images
 *
 *  by Luke Shafer
 *  https://github.com/lukeshafer/parallel_project
 *
 *  nvcc shafer_project.cu -lm -lGL -lGLU -lglut
 */

#include <iostream>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include "hip/hip_runtime.h"


#define XDIM 101
#define YDIM 101
#define RMAX 142

using namespace std;


__global__ void find_radius ( int *in ) {
    
    //int a = blockIdx.x;
    //int b = blockIdx.y;
}

__global__ void hough ( int *in , int *rank) {

    int x = blockIdx.x;
    int y = blockIdx.y;
    if ( in[x+XDIM*y] != 0 ) {
        for (int a = 0; a < XDIM; a++) {
            for (int b = 0; y < YDIM; y++) {
                int i = sqrt( pow(x-a,2) + pow(x-b,2) );
                rank[x+XDIM*y+XDIM*YDIM+i]++;
            }
        }
    }

}

int main( void ) {

    string input_file = "MoonOriginal.png";

    int input[XDIM][YDIM]; // test example
    int *dev_in;

    int rank[XDIM][YDIM][RMAX] = {0};
    int *dev_rank;

    //allocate GPU memory
    hipMalloc( (void**)&dev_in, XDIM * YDIM * sizeof(int) ); 
    hipMalloc( (void**)&dev_rank, XDIM * YDIM * RMAX * sizeof(int) ); 
    
    // Generate circle using CPU. Ideally would be replaced with actual input image
    int cx = round(XDIM/2); // center of circle
    int cy = round(YDIM/2);
    int r = 40; //value must be hard-coded

    ofstream outfile("output.csv");
    if (outfile.is_open()) {

        for (int y=0; y<XDIM; y++) {
            for (int x=0; x<YDIM; x++) {
                if ( round( sqrt( pow((x-cx),2) + pow((y-cx),2) ) ) == r ) {
                    input[x][y] = 255;
                } else {
                    input[x][y] = 0;
                }
                outfile << input[x][y] << ",";
            }
            outfile << "\n";
        }
        outfile.close();
    } else printf("Unable to open file");

    // At this point, we have our circle image as a 2d array of integers
    
    // copy input to GPU memory
    hipMemcpy( dev_in, input, XDIM * YDIM * sizeof(int),
                              hipMemcpyHostToDevice );
    hipMemcpy( dev_rank, rank, XDIM * YDIM * RMAX * sizeof(int),
                              hipMemcpyHostToDevice );

    

    dim3    grid(XDIM,YDIM);
    hough<<<grid,1>>>(dev_in, dev_rank);

    hipFree( dev_in );
    hipFree( dev_rank );

    return 0;
}
